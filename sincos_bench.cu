// sincos_bench.cu
// ----------------------------------------------------------------------------
// CUDA Compute Benchmark for sin/cos and Mandelbrot overlap
// ----------------------------------------------------------------------------

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

// エラーチェックをシンプルな関数に
inline void checkCuda(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::fprintf(stderr,
                     "CUDA error %s (%d) at %s:%d\n",
                     hipGetErrorString(err), int(err), file, line);
        std::exit(EXIT_FAILURE);
    }
}
#define CUDA_CHECK(err) (checkCuda((err), __FILE__, __LINE__))

// モードを列挙型で定義
enum class Mode : int {
    SinMandel = 1,  // sin + Mandelbrot
    SinOnly    = 2,  // sin only
    MandelOnly = 3,  // Mandelbrot only
    CosOnly    = 4   // cos only
};

// マンデルブロー１ステップ
__device__ __forceinline__ float2 mandelbrot_step(float2 z) {
    const float2 c = { -0.7f, 0.27015f };
    // z := (x^2 - y^2, 2xy) + c
    return make_float2(
        z.x * z.x - z.y * z.y + c.x,
        2.0f * z.x * z.y + c.y
    );
}

// ベンチマークカーネル（テンプレートで分岐を解消）
template<Mode M>
__global__ void benchKernel(uint32_t iterations, float* result) {
    const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    float x  = 0.000123f * (gid + 1);
    float2 z = make_float2(0.0001f * (gid / 64), x);

    for (uint32_t i = 0; i < iterations; ++i) {
        if constexpr (M == Mode::SinMandel) {
            #pragma unroll
            for (int j = 0; j < 8; ++j) {
                x = __sinf(x);
                z = mandelbrot_step(z);
            }
        }
        else if constexpr (M == Mode::SinOnly) {
            #pragma unroll
            for (int j = 0; j < 8; ++j) {
                x = __sinf(x);
            }
        }
        else if constexpr (M == Mode::MandelOnly) {
            #pragma unroll
            for (int j = 0; j < 8; ++j) {
                z = mandelbrot_step(z);
            }
        }
        else { // Mode::CosOnly
            x = __cosf(x);
        }
    }

    // ブロック内スレッド 0 が書き込み
    if (threadIdx.x == 0) {
        result[0] = x + z.x;
    }
}

// カーネル起動支援関数
void launch(Mode mode, uint32_t iterations, int device = 0) {
    CUDA_CHECK(hipSetDevice(device));

    // Managed メモリで簡潔に
    float* d_result = nullptr;
    CUDA_CHECK(hipMallocManaged(&d_result, sizeof(float)));
    d_result[0] = 0.0f;

    constexpr int THREADS = 1024;
    constexpr int BLOCKS  = 64;

    switch (mode) {
        case Mode::SinMandel:
            benchKernel<Mode::SinMandel><<<BLOCKS, THREADS>>>(iterations, d_result);
            break;
        case Mode::SinOnly:
            benchKernel<Mode::SinOnly><<<BLOCKS, THREADS>>>(iterations, d_result);
            break;
        case Mode::MandelOnly:
            benchKernel<Mode::MandelOnly><<<BLOCKS, THREADS>>>(iterations, d_result);
            break;
        case Mode::CosOnly:
            benchKernel<Mode::CosOnly><<<1, 1>>>(iterations, d_result); // Cos はシングル
            break;
    }

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    std::printf("Result = %f\n", d_result[0]);
    CUDA_CHECK(hipFree(d_result));
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::printf("Usage: %s <mode 1-4> <iterations> [device]\n", argv[0]);
        std::printf(" 1: Sin+Mandel  2: Sin only  3: Mandel only  4: Cos only\n");
        return 0;
    }

    int m = std::atoi(argv[1]);
    uint32_t iters = std::atoi(argv[2]);
    int dev = (argc >= 4) ? std::atoi(argv[3]) : 0;

    if (m < 1 || m > 4) {
        std::fprintf(stderr, "Invalid mode: %d\n", m);
        return EXIT_FAILURE;
    }

    launch(static_cast<Mode>(m), iters, dev);
    return 0;
}
